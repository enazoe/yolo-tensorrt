#include "hip/hip_runtime.h"
#include <cmath>
#include <stdio.h>
#include <cassert>
#include <iostream>
#include "mish.h"

namespace nvinfer1
{
    MishPlugin::MishPlugin()
    {
    }

    MishPlugin::~MishPlugin()
    {
    }

    // create the plugin at runtime from a byte stream
    MishPlugin::MishPlugin(const void* data, size_t length)
    {
        assert(length == sizeof(input_size_));
        input_size_ = *reinterpret_cast<const int*>(data);
    }

    void MishPlugin::serialize(void* buffer) const noexcept
    {
        *reinterpret_cast<int*>(buffer) = input_size_;
    }

    size_t MishPlugin::getSerializationSize() const noexcept
    {  
        return sizeof(input_size_);
    }

    int MishPlugin::initialize()noexcept
    { 
        return 0;
    }

	bool MishPlugin::supportsFormat(DataType type, PluginFormat format) const noexcept
	{
		return (type == DataType::kFLOAT && format == PluginFormat::kLINEAR);
	}

	void MishPlugin::configureWithFormat(const Dims* inputDims, int nbInputs,
		const Dims* outputDims, int nbOutputs, DataType type, PluginFormat format, int maxBatchSize) noexcept
	{

	}

    Dims MishPlugin::getOutputDimensions(int index, const Dims* inputs, int nbInputDims)noexcept
    {
        assert(nbInputDims == 1);
        assert(index == 0);
        input_size_ = inputs[0].d[0] * inputs[0].d[1] * inputs[0].d[2];
        // Output dimensions
        return Dims3(inputs[0].d[0], inputs[0].d[1], inputs[0].d[2]);
    }

    // Set plugin namespace
    void MishPlugin::setPluginNamespace(const char* pluginNamespace)noexcept
    {
        mPluginNamespace = pluginNamespace;
    }

    const char* MishPlugin::getPluginNamespace() const noexcept
    {
        return mPluginNamespace;
    }

    // Return the DataType of the plugin output at the requested index
    DataType MishPlugin::getOutputDataType(int index, const nvinfer1::DataType* inputTypes, int nbInputs) const noexcept
    {
        return DataType::kFLOAT;
    }

    // Return true if output tensor is broadcast across a batch.
    bool MishPlugin::isOutputBroadcastAcrossBatch(int outputIndex, const bool* inputIsBroadcasted, int nbInputs) const noexcept
    {
        return false;
    }

    // Return true if plugin can use input that is broadcast across batch without replication.
    bool MishPlugin::canBroadcastInputAcrossBatch(int inputIndex) const noexcept
    {
        return false;
    }

    void MishPlugin::configurePlugin(const PluginTensorDesc* in, int nbInput, const PluginTensorDesc* out, int nbOutput)noexcept
    {
    }

    // Attach the plugin object to an execution context and grant the plugin the access to some context resource.
    void MishPlugin::attachToContext(cudnnContext* cudnnContext, cublasContext* cublasContext, IGpuAllocator* gpuAllocator)noexcept
    {
    }

    // Detach the plugin object from its execution context.
    void MishPlugin::detachFromContext()noexcept {}

    const char* MishPlugin::getPluginType() const noexcept
    {
        return "Mish_TRT";
    }

    const char* MishPlugin::getPluginVersion() const noexcept
    {
        return "1";
    }

    void MishPlugin::destroy()noexcept
    {
        delete this;
    }

    // Clone the plugin
    IPluginV2* MishPlugin::clone() const noexcept
    {
        MishPlugin *p = new MishPlugin();
        p->input_size_ = input_size_;
        p->setPluginNamespace(mPluginNamespace);
        return p;
    }

    __device__ float tanh_activate_kernel(float x){return (2/(1 + expf(-2*x)) - 1);}

    __device__ float softplus_kernel(float x, float threshold = 20) 
	{
        if (x > threshold) return x;                // too large
        else if (x < -threshold) return expf(x);    // too small
        return logf(expf(x) + 1);
    }

    __global__ void mish_kernel(const float *input, float *output, int num_elem) 
	{

        int idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= num_elem) return;

        //float t = exp(input[idx]);
        //if (input[idx] > 20.0) {
        //    t *= t;
        //    output[idx] = (t - 1.0) / (t + 1.0);
        //} else {
        //    float tt = t * t;
        //    output[idx] = (tt + 2.0 * t) / (tt + 2.0 * t + 2.0);
        //}
        //output[idx] *= input[idx];
        output[idx] = input[idx] * tanh_activate_kernel(softplus_kernel(input[idx]));
    }

    void MishPlugin::forwardGpu(const float *const * inputs, float* output, hipStream_t stream, int batchSize)
	{
        int block_size = thread_count_;
        int grid_size = (input_size_ * batchSize + block_size - 1) / block_size;
        mish_kernel<<<grid_size, block_size>>>(inputs[0], output, input_size_ * batchSize);
    }

	int MishPlugin::enqueue(int batchSize,
		const void* const* inputs,
		void* const* outputs,
		void* workspace,
		hipStream_t stream) noexcept 
	{
        //assert(batchSize == 1);
        //GPU
        //CUDA_CHECK(hipStreamSynchronize(stream));
        forwardGpu((const float *const *)inputs, (float*)outputs[0], stream, batchSize);
        return 0;
    }

    int MishPlugin::enqueue(int batchSize,
		const void* const* inputs,
		void** outputs,
		void* workspace,
		hipStream_t stream) noexcept 
	{
        return enqueue(batchSize, inputs, (void* const*)outputs, workspace, stream);
    }

    PluginFieldCollection MishPluginCreator::mFC{};
    std::vector<PluginField> MishPluginCreator::mPluginAttributes;

    MishPluginCreator::MishPluginCreator()
    {
        mPluginAttributes.clear();

        mFC.nbFields = mPluginAttributes.size();
        mFC.fields = mPluginAttributes.data();
    }

    const char* MishPluginCreator::getPluginName() const noexcept
    {
            return "Mish_TRT";
    }

    const char* MishPluginCreator::getPluginVersion() const noexcept
    {
            return "1";
    }

    const PluginFieldCollection* MishPluginCreator::getFieldNames()noexcept
    {
            return &mFC;
    }

    IPluginV2* MishPluginCreator::createPlugin(const char* name, const PluginFieldCollection* fc)noexcept
    {
        MishPlugin* obj = new MishPlugin();
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }

    IPluginV2* MishPluginCreator::deserializePlugin(const char* name, const void* serialData, size_t serialLength)noexcept
    {
        // This object will be deleted when the network is destroyed, which will
        // call MishPlugin::destroy()
        MishPlugin* obj = new MishPlugin(serialData, serialLength);
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }


	void MishPluginCreator::setPluginNamespace(const char* libNamespace)noexcept
	{
		mNamespace = libNamespace;
	}

	const char* MishPluginCreator::getPluginNamespace() const noexcept
	{
		return mNamespace.c_str();
	}


}

